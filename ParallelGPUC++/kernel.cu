#include "hip/hip_runtime.h"

#include <stdint.h>
#include "hip/hip_complex.h"

__device__ 
void PaintTheBelongingPoint(uint8_t* &colors, int &width, int &x, int &y)
{
	for (int j = 0; j < 3; j++)
		colors[y * width * 4 + x * 4 + j] = 0;
	colors[y * width * 4 + x * 4 + 3] = 255;
}

__device__ 
void PaintTheNotBelongingPoint(uint8_t* &colors, int &width, int &x, int &y, int &iter, int &calcIterNum, float &invCalcIterNum)
{
	colors[y * width * 4 + x * 4 + 0] =
		(255 - (calcIterNum - iter) * invCalcIterNum * 255.0f);
	colors[y * width * 4 + x * 4 + 2] =
		(255 - (calcIterNum - 0.5f * iter) * invCalcIterNum * 255.0f);
}

__global__ 
void FractalCalcOnDevice(uint8_t* colors, int* width, int* height, float* scale, float* offsetX, float* offsetY, int* calcIterNum)
{
	int _width = *width;
	int _height = *height;
	float _scale = *scale;
	float _offsetX = *offsetX;
	float _offsetY = *offsetY;
	int _calcIterNum = *calcIterNum;
	float invCalcIterNum = 1.0f / (_calcIterNum - 1);
	
	hipFloatComplex c = make_hipComplex(0.0f, 0.0f);
	hipFloatComplex z = make_hipComplex(0.0f, 0.0f);
	hipFloatComplex zPrev = make_hipComplex(hipCrealf(c), hipCimagf(c));

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	float mathX = 0.0;
	float mathY = 0.0;

	int y = index / _width;
	int x = index % _width;
	
	while (y * _width + x < _width * _height)
	{
		mathX = (x - _width / 2 + _offsetX) / _scale;
		mathY = (_height / 2 - y - _offsetY) / _scale;

		//
		// Begin with the 1-st iteration. (Not with 0-th)
		c.x = mathX;
		c.y = mathY;
		z.x = 0.0f;
		z.y = 0.0f;
		zPrev.x = mathX;
		zPrev.y = mathY;

		int iter = 1;
		while (iter < _calcIterNum &&
			z.x * z.x + z.y * z.y < 4.0f)
		{
			z = hipCaddf(hipCmulf(zPrev, zPrev), c);
			zPrev = z;

			iter++;
		}
		if (iter >= _calcIterNum)
		{
			PaintTheBelongingPoint(colors, _width, x, y);
		}
		else
		{
			PaintTheNotBelongingPoint(colors, _width, x, y, iter, _calcIterNum, invCalcIterNum);
		}

		index += stride;
		y = index / _width;
		x = index % _width;
	}
}

extern "C" __declspec(dllexport)
__host__ 
void ParallelGPUFractalCalc(uint8_t* colors, int width, int height, float scale, float offsetX, float offsetY, int calcIterNum)
{
	uint8_t* d_colors;
	hipMalloc(&d_colors, width * height * 4 * sizeof(uint8_t));
	hipMemcpy(d_colors, colors, width * height * 4 * sizeof(uint8_t), hipMemcpyHostToDevice);

	int* d_width;
	hipMalloc(&d_width, sizeof(int));
	hipMemcpy(d_width, &width, sizeof(int), hipMemcpyHostToDevice);

	int* d_height;
	hipMalloc(&d_height, sizeof(int));
	hipMemcpy(d_height, &height, sizeof(int), hipMemcpyHostToDevice);

	float* d_scale;
	hipMalloc(&d_scale, sizeof(float));
	hipMemcpy(d_scale, &scale, sizeof(float), hipMemcpyHostToDevice);

	float* d_offsetX;
	hipMalloc(&d_offsetX, sizeof(float));
	hipMemcpy(d_offsetX, &offsetX, sizeof(float), hipMemcpyHostToDevice);

	float* d_offsetY;
	hipMalloc(&d_offsetY, sizeof(float));
	hipMemcpy(d_offsetY, &offsetY, sizeof(float), hipMemcpyHostToDevice);

	int* d_calcIterNum;
	hipMalloc(&d_calcIterNum, sizeof(int));
	hipMemcpy(d_calcIterNum, &calcIterNum, sizeof(int), hipMemcpyHostToDevice);

	int blockSize = 256;
	int blocksNum = (width * height * 4 + blockSize - 1) / blockSize;
	FractalCalcOnDevice <<< blocksNum, blockSize >>> (d_colors, d_width, d_height, d_scale, d_offsetX, d_offsetY, d_calcIterNum);

	hipMemcpy(colors, d_colors, width * height * 4 * sizeof(uint8_t), hipMemcpyDeviceToHost);

	hipFree(d_calcIterNum);
	hipFree(d_colors);
	hipFree(d_height);
	hipFree(d_offsetX);
	hipFree(d_offsetY);
	hipFree(d_scale);
	hipFree(d_width);
}